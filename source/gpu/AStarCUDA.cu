#include "hip/hip_runtime.h"
/**
 * Parallel implementation of A* Search
 */

#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <float.h>
#include <string.h>
#include "AStarCUDA.h"
#include "ClosedSet.h"
#include "Queue.h"

/**
 * Constants used for node validation
 */
#define AS_V_IDLE		0 /* Node not visited yet */
#define AS_V_VALID		1 /* The node is valid */
#define AS_V_INVALID	2 /* The node is invalid */

#define AS_STATE_AT(base, index) (((char*)base) + index*config->stateSize)

using namespace std;


Queue * newQueue(int capacity, int increaseCapacity){
	Queue * queue = new Queue;
	queue->capacity = capacity+1;
	queue->increaseCapacity = increaseCapacity;
	queue->list = (AS_NodePointer *) malloc(sizeof(AS_NodePointer)*(queue->capacity));
	queue->index = 1;
	return queue;
}

void Queue_insert(Queue * queue, AS_NodePointer node){
	if(queue->index >= queue->capacity){
		queue->list = (AS_NodePointer *) realloc(queue->list, sizeof(AS_NodePointer)*(queue->capacity+AS_QUEUE_INCREASE_CAPACITY));
		if(!queue->list){
			fprintf(stderr, "Error on increasing queue capacity.\n");
			exit(1);			
		}
		queue->capacity += AS_QUEUE_INCREASE_CAPACITY;
	}
	int index = queue->index;
	queue->index++;
	double nodeCost = node->cost + node->heuristic;
	while(index/2 > 0){
		AS_NodePointer parent = queue->list[index/2];
		if(parent->cost + parent->heuristic > nodeCost){
			queue->list[index] = parent;
			index = index/2;
		}else{
			break;
		}
	}
	queue->list[index] = node;
}

inline bool Queue_isEmpty(Queue * queue){
	return queue->index == 1;
}

AS_NodePointer Queue_remove(Queue * queue){
	AS_NodePointer returnValue = queue->list[1];
	queue->index--;
	if(queue->index == 1) return returnValue;
	
	AS_NodePointer rightMost = queue->list[queue->index];
	double rightMostCost = rightMost->cost + rightMost->heuristic;
	int index = 2;
	while(index < queue->index){
		AS_NodePointer left = queue->list[index];
		AS_NodePointer right = index+1 < queue->index ? queue->list[index+1] : NULL;
		double leftCost = left->cost + left->heuristic;
		double rightCost = right == NULL ? DBL_MAX : right->cost + right->heuristic;
		if(rightMostCost > leftCost || rightMostCost > rightCost){
			int selectedIndex = index;
			if(right && rightCost < leftCost){
				selectedIndex++;
			}
			queue->list[index/2] = queue->list[selectedIndex];
			index = 2*selectedIndex;
		}else{
			break;
		}
	}
	queue->list[index/2] = rightMost;
	
	return returnValue;
}

void Queue_free(Queue * queue){
	free(queue->list);
	delete queue;
}

ClosedSet * newClosedSet(bool (* areSameStates)(void * stateA, void * stateB), int chunkSize = AS_CLOSEDSET_CHUNK_SIZE){
	ClosedSet * closedSet = new ClosedSet;
	closedSet->list = new ClosedSetList;
	closedSet->currentList = closedSet->list;
	closedSet->chunkSize = chunkSize;
	closedSet->length = 0;
	closedSet->areSameStates = areSameStates;
	
	closedSet->list->nodes = new AS_NodePointer[chunkSize];
	closedSet->list->next = NULL;
	return closedSet;
}

void ClosedSet_freeList(ClosedSetList * list){
	if(list->next) ClosedSet_freeList(list->next);
	delete [] list->nodes;
	delete list;
}

void ClosedSet_free(ClosedSet * closedSet){
	ClosedSet_freeList(closedSet->list);
	delete closedSet;
}

void ClosedSet_add(ClosedSet * closedSet, AS_NodePointer node){
	if(closedSet->length == 0){
		closedSet->currentList->nodes[0] = node;
	}else{
		int index = closedSet->length % closedSet->chunkSize;
		if(index == 0){
			closedSet->currentList->next = new ClosedSetList;
			closedSet->currentList = closedSet->currentList->next;
			closedSet->currentList->nodes = new AS_NodePointer[closedSet->chunkSize];
			closedSet->currentList->next = NULL;
		}
		closedSet->currentList->nodes[index] = node;
	}
	closedSet->length++;
}

bool ClosedSet_hasNode(ClosedSet * closedSet, AS_NodePointer node){
	return ClosedSet_hasState(closedSet, node->state);
}

bool ClosedSet_hasState(ClosedSet * closedSet, void * state){
	ClosedSetList * list = closedSet->list;
	int chunkSize = closedSet->chunkSize;
	int length = closedSet->length;
	int count = 0;
	while(list){
		for(int i = 0; i<chunkSize && count<length; i++, count++){
			if(closedSet->areSameStates(state, list->nodes[i]->state)){
				return true;
			}
		}
		list = list->next;
	}
	return false;
}

void AS_initConfig(AS_Config * config){
	config->closedSetChunkSize = AS_CLOSEDSET_CHUNK_SIZE;
	config->queueInitialCapacity = AS_QUEUE_INITIAL_CAPACITY;
	config->queueIncreaseCapacity = AS_QUEUE_INCREASE_CAPACITY;
	config->nodesPerCycle = AS_NODES_PER_CYCLE;
	config->maxNodesPerExpansion = AS_MAX_NODES_PER_EXPANSION;
	config->dataSize = sizeof(int);
	config->dataSize = 0;
}

void AS_freeTree(AS_Node * root, AS_Config * config){
	if(root){
		for(int i = 0; i<root->childrenLength; i++){
			AS_freeTree(root->children[i], config);
		}
		root->childrenLength = 0;
		if(root->status == AS_STATUS_IDLE){ //If it is not in the path
			ASNode_free(root, config);
		}
	}
}

AS_NodePointer * AS_searchResult(AS_Node * node, AS_Config * config){
	/* Count the number of nodes */
	AS_Node * n = node;
	int count = 0;
	while(n){
		count++;
		n = n->parent;
	}
	
	/* Creating array and filling it */
	AS_NodePointer * path = new AS_NodePointer[count+1];
	path[count] = NULL;
	count--;
	n = node;
	while(true){
		path[count] = n;
		n->status = AS_STATUS_IN_PATH;
		if(n->parent){
			n = n->parent;
			count--;
		}else{
			break;
		}
	}
	
	AS_freeTree(n, config);
	
	return path;
}

__global__ void AS_nodeCycle(ExpandStateFunction expandState, int stateSize, void * statesToExpand_d, void * expansionStates_d, NodeData * expansionNodesData_d, int * expansionSizes_d){
	void * state = ((char *)statesToExpand_d) + blockIdx.x*stateSize;
	void * expansionStates = ((char*)expansionStates_d) + blockIdx.x*blockDim.x*stateSize;
	int * expansionLength = expansionSizes_d + blockIdx.x;
	NodeData * nodesData = expansionNodesData_d + blockIdx.x*blockDim.x;
	expandState(state, expansionStates, nodesData, expansionLength);
}

void statesToNodes(void * states, AS_Node * nodes, int count, size_t stateSize){
	for(int i = 0; i<count; i++){
		nodes[i].state = states;
		states = ((char *)states) + stateSize;
	}
}

AS_NodePointer * AS_search(AS_Config * config){
	AS_NodePointer * path = NULL;
	
	/* Allocate space for expansion */
	int expansionNodesLength = config->nodesPerCycle * (config->maxNodesPerExpansion);
	
	/* Vector with all the states to be used by the device on expansion */
	void * expansionStates_d;
	hipMalloc((void **) &expansionStates_d, expansionNodesLength * config->stateSize);
	/* Vector with all the node data structs to be used by the device on expansion */
	NodeData * expansionNodesData_d;
	hipMalloc((void **) &expansionNodesData_d, expansionNodesLength * sizeof(NodeData));
	/* Vector with size of each expansion */
	int * expansionSizes_d;
	hipMalloc((void **) &expansionSizes_d, sizeof(int) * config->nodesPerCycle);
	/* States to be expanded to be used by the device */
	void * statesToExpand_d;
	hipMalloc((void **) &statesToExpand_d, config->stateSize * config->nodesPerCycle);

	/* The host version of the variables above */
	void * expansionStates_h =  malloc(expansionNodesLength * config->stateSize);
	NodeData * expansionNodesData_h = (NodeData *) malloc(sizeof(NodeData) * expansionNodesLength);
	int * expansionSizes_h = (int *) malloc(sizeof(int) * config->nodesPerCycle);
	void * statesToExpand_h = malloc(config->stateSize * config->nodesPerCycle);
	
	/* Used to solve conflicts due to the parallel expansion of nodes */
	char * expansionValidity = (char *) malloc(sizeof(char) * expansionNodesLength);

	AS_NodePointer * nodesToExpand = (AS_NodePointer *) malloc(sizeof(AS_NodePointer) * config->nodesPerCycle);

	ClosedSet * closedSet = newClosedSet(config->areSameStates, config->closedSetChunkSize);
	Queue * queue = newQueue(config->queueInitialCapacity, config->queueIncreaseCapacity);
	
	Queue_insert(queue, config->startNode);
	
	int loopCount = 0;
	while(true){
		loopCount++;
		if(Queue_isEmpty(queue)){
			AS_freeTree(config->startNode, config);
			break;
		}
		AS_Node * node = Queue_remove(queue);
		nodesToExpand[0] = node;

		hipMemcpy(statesToExpand_d, &node->state, config->stateSize, hipMemcpyHostToDevice);

		ClosedSet_add(closedSet, node);
		
		/* If first element is the goal state */
		if(config->isGoalState(node->state)){
			path = AS_searchResult(node, config);
			break;
		}
		
		int nodeCount = 1;
		while(nodeCount < config->nodesPerCycle && !Queue_isEmpty(queue)){
			node = Queue_remove(queue);
			nodesToExpand[nodeCount] = node;
			hipMemcpy(AS_STATE_AT(statesToExpand_d, nodeCount), &node->state, config->stateSize,  hipMemcpyHostToDevice);
			
			ClosedSet_add(closedSet, node);
			nodeCount++;
		}


		
		/* PARALLELISM */
		AS_nodeCycle<<< nodeCount, config->maxNodesPerExpansion >>>(config->expandState, config->stateSize, statesToExpand_d, expansionStates_d, expansionNodesData_d, expansionSizes_d);
		/* Copy data to host */
		hipMemcpy(expansionStates_h, expansionStates_d, config->stateSize * expansionNodesLength, hipMemcpyDeviceToHost);
		hipMemcpy(expansionNodesData_h, expansionNodesData_d, sizeof(NodeData) * expansionNodesLength, hipMemcpyDeviceToHost);
		hipMemcpy(expansionSizes_h, expansionSizes_d, sizeof(int)*nodeCount, hipMemcpyDeviceToHost);
		
		
		/* 
		 * Mark which nodes in the expansion are valid, i.e., for repeated states,
		 * the state with the minimal cost is set as valid, the remainder as invalid.
		 * PS: Maybe we can make this parallel too.
		 */
		memset(expansionValidity, AS_V_IDLE, expansionNodesLength);
		/* li - localIndex, bi - blockIndex */
		for(int i = 0, bi=0, li=0; i<expansionNodesLength; ){
			if(li < expansionSizes_h[bi] && expansionValidity[i] == AS_V_IDLE){
				expansionValidity[i] = AS_V_VALID;
				int validIndex = i;
				double validCost = expansionNodesData_h[i].cost + expansionNodesData_h[i].heuristic;
				/* lj - localIndex, bj - blockIndex */
				for(int bj = bi+1, lj = 0, j = bj*config->maxNodesPerExpansion; j<expansionNodesLength; ){
					bool goToNextBlock = false;
					if(lj < expansionSizes_h[bj] && expansionValidity[j] == AS_V_IDLE){
						void * stateA = AS_STATE_AT(expansionStates_h, validIndex);
						void * stateB = AS_STATE_AT(expansionStates_h, j);
						if(config->areSameStates(stateA, stateB)){
							double costB = expansionNodesData_h[j].cost + expansionNodesData_h[j].heuristic;
							if(costB < validCost){
								expansionValidity[j] = AS_V_VALID;
								expansionValidity[validIndex] = AS_V_INVALID;
								validCost = costB;
								validIndex = j;
							}else{
								expansionValidity[j] = AS_V_INVALID;
							}
							goToNextBlock = true;
						}
					
					}
					if(lj + 1 < expansionSizes_h[bj] && !goToNextBlock){
						j++;
						lj++;
					}else if(bj + 1 < nodeCount){
						bj++;
						j = bj * config->maxNodesPerExpansion;
						lj = 0;
					}else{
						break;
					}
				}
			}
			if(li + 1 < expansionSizes_h[bi]){
				i++;
				li++;
			}else if(bi + 1 < nodeCount){
				bi++;
				i = bi * config->maxNodesPerExpansion;
				li = 0;
			}else{
				break;
			}
		}
		
		/*Creating the nodes and adding the children */
		for(int i = 0; i<nodeCount; i++){
			void * childStates = AS_STATE_AT(expansionStates_h, config->maxNodesPerExpansion*i);
			int size = expansionSizes_h[i];
			char * validity = expansionValidity + config->maxNodesPerExpansion*i;
			NodeData * nodeData = expansionNodesData_h + config->maxNodesPerExpansion*i;
			int j = 0;
			AS_Node * node = nodesToExpand[i];
			node->children = (AS_NodePointer *) malloc(sizeof(AS_NodePointer) * config->maxNodesPerExpansion);
			int c = 0; /* Counter for actual children */
			while(j<size){
				if(validity[j] && !ClosedSet_hasState(closedSet, AS_STATE_AT(childStates, j))){
					/* Create the child node */
					void * s = malloc(config->stateSize);
					memcpy(s, AS_STATE_AT(childStates, j), config->stateSize);
					AS_Node * child = newASNode(s, nodeData[j].heuristic, nodeData[j].cost, node);
					node->children[c] = child;
					c++;
					Queue_insert(queue, child);
				}
				j++;
			}
			node->childrenLength = c;
			node->children = (AS_NodePointer *) realloc(node->children, c*sizeof(AS_NodePointer));
		}
	}
	
	printf("loop count = %d\n", loopCount);
	
	Queue_free(queue);
	ClosedSet_free(closedSet);
	
	hipFree(expansionStates_d);
	hipFree(expansionNodesData_d);
	hipFree(expansionSizes_d);
	hipFree(statesToExpand_d);

	free(expansionStates_h);
	free(expansionNodesData_h);
	free(expansionSizes_h);
	free(statesToExpand_h);
	free(expansionValidity);
	free(nodesToExpand);
	
	return path;
}

void AS_freePath(AS_NodePointer * path, AS_Config * config){
	for(int i = 0; path[i]; i++){
		ASNode_free(path[i], config);
	}
	delete [] path;
}

AS_Node * newASNode(void * state, double heuristic, double cost, AS_Node * parent){
	AS_Node * node = (AS_Node *) malloc(sizeof(AS_Node));
	ASNode_init(node, state, heuristic, cost, parent);
	return node;
}

__host__ __device__ void ASNode_init(AS_Node * node, void * state, double heuristic, double cost, AS_Node * parent){
	node->data = NULL;
	node->state = state;
	node->parent = parent;
	node->heuristic = heuristic;
	node->cost = cost;
	node->status = AS_STATUS_IDLE;
	node->childrenLength = 0;
	node->children = NULL;
}

void ASNode_free(AS_Node * node, AS_Config * config){
	if(node->state){
		free(node->state);
		node->state = NULL;
	}
	if(node->data){
		free(node->data);
		node->data = NULL;
	}
	free(node->children);
	free(node);
}

void testQueue(){
	Queue * q = newQueue(1000, 100);
	
	printf("Testing with only one element:\n");
	printf("\tInserting...\n");
	Queue_insert(q, newASNode(NULL, 1));
	printf("\tRemoving...\n");
	AS_Node * n = Queue_remove(q);
	printf("\tHeuristic: %f\n\n", n->heuristic);
	delete n;
	
	printf("Testing with 2 elements:\n");
	Queue_insert(q, newASNode(NULL, 2));
	Queue_insert(q, newASNode(NULL, 1));
	printf("\tExpected output: 1 2\n\tRemoving elements and printing values:\n");
	n = Queue_remove(q);
	printf("\t%f ", n->heuristic);
	delete n;
	n = Queue_remove(q);
	printf("%f\n\n ", n->heuristic);
	
	
	printf("Testing with 5 elements with random heuristics:\n\tInput order (heuristic values):");
	for(int i = 0; i<5; i++){
		int h = rand() % 100;
		printf(" %d", h);
		Queue_insert(q, newASNode(NULL, h));
	}
	printf("\n\tOutput:");
	for(int i = 0; i<5; i++){
		n = Queue_remove(q);
		printf(" %d", (int) n->heuristic);
		delete n;
	}
	printf("\n\tQueue is empty: %s", Queue_isEmpty(q) ? "YES" : "NO");
	
	Queue_free(q);
}

bool testClosedSet_isSameState(void * stateA, void * stateB){
	int a = *((int *) stateA);
	int b = *((int *) stateB);
	return a == b;
}
void testClosedSet(){
	ClosedSet * cs = newClosedSet(&testClosedSet_isSameState);
	AS_NodePointer n = newASNode();
	int * x = new int;
	*x = 1;
	n->state = x;
	
	printf("Adding only one element...\n");
	ClosedSet_add(cs, n);
	bool hasNode = ClosedSet_hasNode(cs, n);
	printf("\tHas node (expected YES): %s\n", hasNode ? "YES" : "NO");
	
	printf("Adding more 5 elements...\n");
	for(int i = 0; i<5; i++){
		AS_NodePointer n = newASNode();
		int * x = new int;
		*x = rand();
		n->state = x;
		ClosedSet_add(cs, n);
	}
	
	printf("Adding more CHUNK SIZE number of elements...\n");
	for(int i = 0; i<AS_CLOSEDSET_CHUNK_SIZE; i++){
		n = newASNode();
		x = new int;
		*x = rand();
		n->state = x;
		ClosedSet_add(cs, n);
	}
	printf("Verifying if the closed set has the last element added...\n");
	hasNode = ClosedSet_hasNode(cs, n);
	printf("\tHas node (expected YES): %s\n", hasNode ? "YES" : "NO");
	
	ClosedSet_free(cs);
}
